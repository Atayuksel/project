#include "hip/hip_runtime.h"
#include <Python.h>
#include <mkl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <numpy/arrayobject.h>
#include "hipblas.h"

#define SEED    1
#define BRNG    VSL_BRNG_MCG31
#define METHOD  0

__global__
static void _trainNetwork(int sizeDataset, int* source_idx, int*target_idx, int lexiconSize,
	int hiddenUnitCount, double* hiddenWeights, double* out){

	double *varHiddenUnits, *varOutputUnits;
	hipMalloc(&varHiddenUnits, (hiddenUnitCount * sizeof(double)));
	hipMalloc(&varOutputUnits, (lexiconSize * sizeof(double)));

	double learningFactor = 0.1;

	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	printf("Size dataset is %d\n", sizeDataset);

	for (int idx = 0; idx < sizeDataset; idx++) {
		int sourceWordIdx = source_idx[idx];
		int targetWordIdx = target_idx[idx];

		// obtain values of hidden units to varHiddenUnits
		double *hiddenWeightsStartPointer = hiddenWeights + (sourceWordIdx * hiddenUnitCount);
		memcpy(varHiddenUnits, hiddenWeightsStartPointer, (hiddenUnitCount * sizeof(double)));

		printf("[ ");
		for (int i = 0; i < hiddenUnitCount; i++) {
			printf("%f, ", varHiddenUnits[i]);
		}
		printf(" ]\n");

		// obtain values of output units via dgemm
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasDgemm(handle,
			HIPBLAS_OP_N, HIPBLAS_OP_T,
			1, lexiconSize, hiddenUnitCount,
			alpha,
			varHiddenUnits, 1,
			out, hiddenUnitCount,
			beta,
			varOutputUnits, 1);

		hipDeviceSynchronize();

		/*printf("[ ");
		for (int i = 0; i < lexiconSize; i++) {
			printf("%f, ", varOutputUnits[i]);
		}
		printf(" ]\n");*/

		double sumOutputUnits = 0.0;
		for (int i = 0; i < lexiconSize; i++) {
			sumOutputUnits = sumOutputUnits + varOutputUnits[i];
		}

		double valScal = (double)1 / sumOutputUnits;
		hipblasDscal(handle, lexiconSize, &valScal, varOutputUnits, 1);

		// calculation update values for output layer weights
		double *upOutWeights, *tmp, *IdxUpWeight;
		hipMalloc(&upOutWeights, (hiddenUnitCount * lexiconSize * sizeof(double)));
		hipMalloc(&tmp, (1 * lexiconSize * sizeof(double)));
		double inv = -1;
		double mult;
		double zero = 0;
		for (int i = 0; i < hiddenUnitCount; i++) {
			memcpy(tmp, varOutputUnits, (lexiconSize * sizeof(double)));
			hipblasDscal(handle, lexiconSize, &inv, tmp, 1);
			tmp[targetWordIdx] = tmp[targetWordIdx] + 1;
			mult = learningFactor * varHiddenUnits[i];
			hipblasDscal(handle, lexiconSize,&mult, tmp, 1);
			IdxUpWeight = upOutWeights + i * lexiconSize;
			memcpy(IdxUpWeight, tmp, (lexiconSize * sizeof(double)));
		}

		// calculation update values for hidden layer weights
		double *UpHidWeights, *tmpS, *IdxPointer;
		hipMalloc(&UpHidWeights, (hiddenUnitCount * lexiconSize * sizeof(double)));
		hipMalloc(&tmpS, (lexiconSize * sizeof(double)));
		for (int i = 0; i < hiddenUnitCount; i++) {
			IdxPointer = i * lexiconSize + out;
			memcpy(tmpS, IdxPointer, (lexiconSize * sizeof(double)));
			memcpy(tmp, varOutputUnits, (lexiconSize * sizeof(double)));
			hipblasDscal(handle, lexiconSize, &inv, tmp, 1);
			tmp[targetWordIdx] = tmp[targetWordIdx] + 1;
			for (int j = 0; j < lexiconSize; j++) {
				tmp[j] = tmp[j] * tmpS[j];
			}
			hipblasDscal(handle, lexiconSize, &zero, tmpS, 1);
			tmpS[sourceWordIdx] = tmp[sourceWordIdx] * learningFactor;
			IdxPointer = i * lexiconSize + UpHidWeights;
			memcpy(IdxPointer, tmpS, (lexiconSize * sizeof(double)));
		}
		int hiddenWeightIdx;
		int hiddenUnitIdx = 0;
		int lexiconIdx = 0;
		for (int i = 0; i < lexiconSize * hiddenUnitCount; i++) {
			hiddenUnitIdx = i / lexiconSize;
			lexiconIdx = i - (lexiconSize * hiddenUnitIdx);
			hiddenWeightIdx = (hiddenUnitCount * lexiconIdx) + hiddenUnitIdx;
			hiddenWeights[hiddenWeightIdx] = hiddenWeights[hiddenWeightIdx] + UpHidWeights[i];
			out[i] = out[i] + upOutWeights[i];
		}
		// printf("%d", idx);
	}
}
static PyObject* trainNetwork(PyObject* self, PyObject* args)
{
	PyArrayObject *arr_np_source, *arr_np_target;
	int lexiconSize, numberHiddenUnit;
	int *cpu_arr_source, *cpu_arr_target;

	if (!PyArg_ParseTuple(args, "OOii", &arr_np_source, &arr_np_target, &lexiconSize, &numberHiddenUnit)) {
		return NULL;
	}

	int sizeTrainingSet = arr_np_source->dimensions[0];
	printf("Size of the dataset is %d \n", sizeTrainingSet);

	cpu_arr_source = (int*)malloc(sizeTrainingSet * sizeof(int));
	cpu_arr_target = (int*)malloc(sizeTrainingSet * sizeof(int));

	for (int i = 0; i < sizeTrainingSet; i++) {
		int *item_source;
		int *item_target;
		item_source = (int *)PyArray_GETPTR1(arr_np_source, i);
		item_target = (int *)PyArray_GETPTR1(arr_np_target, i);
		cpu_arr_source[i] = *item_source;
		cpu_arr_target[i] = *item_target;
	}

	// allocating and copying idx of word pairs to gpu memory
	int *gpu_arr_source, *gpu_arr_target;
	hipMalloc(&gpu_arr_source, (sizeTrainingSet * sizeof(int)));
	hipMalloc(&gpu_arr_target, (sizeTrainingSet * sizeof(int)));
	hipMemcpy(gpu_arr_source, cpu_arr_source, (sizeTrainingSet * sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr_target, cpu_arr_target, (sizeTrainingSet * sizeof(int)), hipMemcpyHostToDevice);

	printf("Lexicon size is %d \n", lexiconSize);
	printf("Number of hidden unit is %d \n", numberHiddenUnit);

	// generate random doubles between -0.01 and 0.01 with MKL vdRngUniform
	int N = numberHiddenUnit * lexiconSize;
	double *randomNumsa = (double*)malloc(N * sizeof(double));
	double *randomNumsb = (double*)malloc(N * sizeof(double));
	VSLStreamStatePtr stream;
	double a = -0.01, b = 0.01;
	vslNewStream(&stream, BRNG, SEED);
	vdRngUniform(METHOD, stream, N, randomNumsa, a, b);
	vdRngUniform(METHOD, stream, N, randomNumsb, a, b);
	vslDeleteStream(&stream);

	// allocate and copy random generated weights to gpu
	double *cudaRandA;
	double *cudaRandB;
	hipMalloc(&cudaRandA, (N * sizeof(double)));
	hipMalloc(&cudaRandB, (N * sizeof(double)));
	hipError_t errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	hipMemcpy(cudaRandA, randomNumsa, (N * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(cudaRandB, randomNumsb, (N * sizeof(double)), hipMemcpyHostToDevice);
	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));


	// print weights before kernel
	/*printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsa[i]);
	}
	printf(" ]\n");
	printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsb[i]);
	}
	printf(" ]\n");*/

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	// calling kernel function in GPU
	//add<<<1,1>>>(N, cudaRandA, cudaRandB);
	_trainNetwork<<<1,1>>>(sizeTrainingSet, gpu_arr_source, gpu_arr_target, lexiconSize,
		numberHiddenUnit, cudaRandA, cudaRandB);

	hipDeviceSynchronize();

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	hipMemcpy(randomNumsa, cudaRandA, (N * sizeof(double)), hipMemcpyDeviceToHost);
	hipMemcpy(randomNumsb, cudaRandB, (N * sizeof(double)), hipMemcpyDeviceToHost);

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));


	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	/*printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsa[i]);
	}
	printf(" ]\n");

	printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsb[i]);
	}
	printf(" ]\n");*/

	hipFree(cudaRandA);
	hipFree(cudaRandB);

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	return Py_None;
}

static PyMethodDef word2vecMethods[] = {
	{ "trainNetwork", trainNetwork, METH_VARARGS, "start training of network" },
{ NULL, NULL, 0, NULL }
};

static struct PyModuleDef wordtovec = {
	PyModuleDef_HEAD_INIT,
	"word2vec",
	"word2vec_doc",
	-1,
	word2vecMethods
};

PyMODINIT_FUNC PyInit_wordtovec(void)
{
	PyObject *m;
	m = PyModule_Create(&wordtovec);
	import_array();
	if (m == NULL) {
		return NULL;
	}
	return m;
}
