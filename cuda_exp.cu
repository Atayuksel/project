#include "hip/hip_runtime.h"
#include <Python.h>
#include <mkl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <numpy/arrayobject.h>
#include "hipblas.h"

#define SEED    1
#define BRNG    VSL_BRNG_MCG31
#define METHOD  0

__global__
void bpoutput(double* outputUnitValues, double* hiddenUnitValues, \
								int lexiconSize, double* updateWeightArr, \
								int targetWordIdx) {

	int i = threadIdx.x;

	double *tmp;

	hipblasHandle_t cubpout;
	hipblasCreate(&cubpout);

	hipError_t errSync = hipMalloc(&tmp, (1 * lexiconSize * sizeof(double)));
	printf("tmp allocation in thread %d: %s\n",i, hipGetErrorString(errSync));

	memcpy(tmp, outputUnitValues, (lexiconSize * sizeof(double)));

	double inv = -1;
	hipblasDscal(cubpout, lexiconSize, &inv, tmp, 1);

	tmp[targetWordIdx] = tmp[targetWordIdx] + 1;
	double mult = 0.1 * hiddenUnitValues[i];

	hipblasDscal(cubpout, lexiconSize, &mult, tmp, 1);

	double *IdxUpWeight = updateWeightArr + i * lexiconSize;

	memcpy(IdxUpWeight, tmp, (lexiconSize * sizeof(double)));
	printf("Exiting threadID: %d\n", i);
}

__global__
static void _trainNetwork(int sizeDataset, int* source_idx, int*target_idx, int lexiconSize,
	int hiddenUnitCount, double* hiddenWeights, double* out){

	double *varHiddenUnits, *varOutputUnits;
	hipMalloc(&varHiddenUnits, (hiddenUnitCount * sizeof(double)));
	hipMalloc(&varOutputUnits, (lexiconSize * sizeof(double)));

	double learningFactor = 0.1;

	const double alf = 1;
	const double bet = 0;
	const double *alpha = &alf;
	const double *beta = &bet;

	printf("Size dataset is %d\n", sizeDataset);

	for (int idx = 0; idx < 1; idx++) {
		int sourceWordIdx = source_idx[idx];
		int targetWordIdx = target_idx[idx];

		// obtain values of hidden units to varHiddenUnits
		double *hiddenWeightsStartPointer = hiddenWeights + (sourceWordIdx * hiddenUnitCount);
		memcpy(varHiddenUnits, hiddenWeightsStartPointer, (hiddenUnitCount * sizeof(double)));

		printf("[ ");
		for (int i = 0; i < hiddenUnitCount; i++) {
			printf("%f, ", varHiddenUnits[i]);
		}
		printf(" ]\n");

		// obtain values of output units via dgemm
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasDgemm(handle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			lexiconSize, 1, hiddenUnitCount,
			alpha,
			out, lexiconSize,
			varHiddenUnits, hiddenUnitCount,
			beta,
			varOutputUnits, lexiconSize);

		hipDeviceSynchronize();

		hipError_t errSync = hipGetLastError();
		printf("After dgemm function: %s\n", hipGetErrorString(errSync));

		/*printf("[ ");
		for (int i = 0; i < lexiconSize; i++) {
			printf("%f, ", varOutputUnits[i]);
		}
		printf(" ]\n");*/

		double sumOutputUnits = 0.0;
		for (int i = 0; i < lexiconSize; i++) {
			sumOutputUnits = sumOutputUnits + varOutputUnits[i];
		}

		double valScal = (double)1 / sumOutputUnits;
		hipblasDscal(handle, lexiconSize, &valScal, varOutputUnits, 1);

		errSync = hipGetLastError();
		printf("After dscal func: %s\n", hipGetErrorString(errSync));

		// calculation update values for output layer weights
		double *upOutWeights, *IdxUpWeight;

		errSync = hipMalloc(&upOutWeights, (hiddenUnitCount * lexiconSize * sizeof(double)));
		printf("after cudamalloc: %s\n", hipGetErrorString(errSync));

		double inv = -1;
		double mult;
		double zero = 0;

		bpoutput<<<1,100>>>(varOutputUnits, varHiddenUnits, lexiconSize, \
												upOutWeights, targetWordIdx);

		hipDeviceSynchronize();

		// printf("[ ");
		// for (int i = 0; i < lexiconSize; i++) {
		// 	printf("%f, ", tmp[i]);
		// }
		// printf(" ]\n");

		// calculation update values for hidden layer weights
		double *UpHidWeights, *tmp, *tmpS, *IdxPointer;
		hipMalloc(&UpHidWeights, (hiddenUnitCount * lexiconSize * sizeof(double)));
		hipMalloc(&tmpS, (lexiconSize * sizeof(double)));
		hipMalloc(&tmp, (lexiconSize * sizeof(double)));
		for (int i = 0; i < hiddenUnitCount; i++) {
			IdxPointer = i * lexiconSize + out;
			memcpy(tmpS, IdxPointer, (lexiconSize * sizeof(double)));
			memcpy(tmp, varOutputUnits, (lexiconSize * sizeof(double)));
			hipblasDscal(handle, lexiconSize, &inv, tmp, 1);
			tmp[targetWordIdx] = tmp[targetWordIdx] + 1;
			for (int j = 0; j < lexiconSize; j++) {
				tmp[j] = tmp[j] * tmpS[j];
			}
			hipblasDscal(handle, lexiconSize, &zero, tmpS, 1);
			tmpS[sourceWordIdx] = tmp[sourceWordIdx] * learningFactor;
			IdxPointer = i * lexiconSize + UpHidWeights;
			memcpy(IdxPointer, tmpS, (lexiconSize * sizeof(double)));
		}

		errSync = hipGetLastError();
		printf("After calculation update values for output weights: %s\n", hipGetErrorString(errSync));

		int hiddenWeightIdx;
		int hiddenUnitIdx = 0;
		int lexiconIdx = 0;
		for (int i = 0; i < lexiconSize * hiddenUnitCount; i++) {
			hiddenUnitIdx = i / lexiconSize;
			lexiconIdx = i - (lexiconSize * hiddenUnitIdx);
			hiddenWeightIdx = (hiddenUnitCount * lexiconIdx) + hiddenUnitIdx;
			hiddenWeights[hiddenWeightIdx] = hiddenWeights[hiddenWeightIdx] + UpHidWeights[i];
			out[i] = out[i] + upOutWeights[i];
		}
		errSync = hipGetLastError();
		printf("After calculating hidden weights: %s\n", hipGetErrorString(errSync));
		// printf("%d", idx);

	}
}
static PyObject* trainNetwork(PyObject* self, PyObject* args)
{
	PyArrayObject *arr_np_source, *arr_np_target;
	int lexiconSize, numberHiddenUnit;
	int *cpu_arr_source, *cpu_arr_target;

	if (!PyArg_ParseTuple(args, "OOii", &arr_np_source, &arr_np_target, &lexiconSize, &numberHiddenUnit)) {
		return NULL;
	}

	int sizeTrainingSet = arr_np_source->dimensions[0];
	printf("Size of the dataset is %d \n", sizeTrainingSet);

	cpu_arr_source = (int*)malloc(sizeTrainingSet * sizeof(int));
	cpu_arr_target = (int*)malloc(sizeTrainingSet * sizeof(int));

	for (int i = 0; i < sizeTrainingSet; i++) {
		int *item_source;
		int *item_target;
		item_source = (int *)PyArray_GETPTR1(arr_np_source, i);
		item_target = (int *)PyArray_GETPTR1(arr_np_target, i);
		cpu_arr_source[i] = *item_source;
		cpu_arr_target[i] = *item_target;
	}

	// allocating and copying idx of word pairs to gpu memory
	int *gpu_arr_source, *gpu_arr_target;
	hipMalloc(&gpu_arr_source, (sizeTrainingSet * sizeof(int)));
	hipMalloc(&gpu_arr_target, (sizeTrainingSet * sizeof(int)));
	hipMemcpy(gpu_arr_source, cpu_arr_source, (sizeTrainingSet * sizeof(int)), hipMemcpyHostToDevice);
	hipMemcpy(gpu_arr_target, cpu_arr_target, (sizeTrainingSet * sizeof(int)), hipMemcpyHostToDevice);

	printf("Lexicon size is %d \n", lexiconSize);
	printf("Number of hidden unit is %d \n", numberHiddenUnit);

	// generate random doubles between -0.01 and 0.01 with MKL vdRngUniform
	int N = numberHiddenUnit * lexiconSize;
	double *randomNumsa = (double*)malloc(N * sizeof(double));
	double *randomNumsb = (double*)malloc(N * sizeof(double));
	VSLStreamStatePtr stream;
	double a = -0.01, b = 0.01;
	vslNewStream(&stream, BRNG, SEED);
	vdRngUniform(METHOD, stream, N, randomNumsa, a, b);
	vdRngUniform(METHOD, stream, N, randomNumsb, a, b);
	vslDeleteStream(&stream);

	// allocate and copy random generated weights to gpu
	double *cudaRandA;
	double *cudaRandB;
	hipMalloc(&cudaRandA, (N * sizeof(double)));
	hipMalloc(&cudaRandB, (N * sizeof(double)));
	hipError_t errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	hipMemcpy(cudaRandA, randomNumsa, (N * sizeof(double)), hipMemcpyHostToDevice);
	hipMemcpy(cudaRandB, randomNumsb, (N * sizeof(double)), hipMemcpyHostToDevice);
	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));


	// print weights before kernel
	/*printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsa[i]);
	}
	printf(" ]\n");
	printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsb[i]);
	}
	printf(" ]\n");*/

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	size_t empty, total;
	hipSetDevice(0);
	hipMemGetInfo(&empty, &total);
	printf("%d %d\n",empty/1024,total/1024);

	hipLimit_t limit = hipLimitMallocHeapSize ;
	errSync = hipDeviceGetLimit(&total, limit);
	printf("cuda device get limit: %s\n", hipGetErrorString(errSync));
	printf("Device Limit: %d\n", total/1024);
	errSync = hipDeviceSetLimit(limit, total*10);
	printf("cuda set device limit %s\n", errSync);
	printf("new device limit is:%d\n", total*10/1024);

	clock_t begin = clock();
	_trainNetwork<<<1,1>>>(sizeTrainingSet, gpu_arr_source, gpu_arr_target, lexiconSize,
		numberHiddenUnit, cudaRandA, cudaRandB);


	hipDeviceSynchronize();
	clock_t end = clock();
	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	hipMemcpy(randomNumsa, cudaRandA, (N * sizeof(double)), hipMemcpyDeviceToHost);
	hipMemcpy(randomNumsb, cudaRandB, (N * sizeof(double)), hipMemcpyDeviceToHost);

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));


	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	/*printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsa[i]);
	}
	printf(" ]\n");

	printf("[ ");
	for (int i = 0; i < N; i++) {
		printf("%f, ", randomNumsb[i]);
	}
	printf(" ]\n");*/

	hipFree(cudaRandA);
	hipFree(cudaRandB);

	errSync = hipGetLastError();
	printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Time consumed for a single iter: %f\n", time_spent);
	return Py_None;
}

static PyMethodDef word2vecMethods[] = {
	{ "trainNetwork", trainNetwork, METH_VARARGS, "start training of network" },
{ NULL, NULL, 0, NULL }
};

static struct PyModuleDef wordtovec = {
	PyModuleDef_HEAD_INIT,
	"word2vec",
	"word2vec_doc",
	-1,
	word2vecMethods
};

PyMODINIT_FUNC PyInit_wordtovec(void)
{
	PyObject *m;
	m = PyModule_Create(&wordtovec);
	import_array();
	if (m == NULL) {
		return NULL;
	}
	return m;
}
